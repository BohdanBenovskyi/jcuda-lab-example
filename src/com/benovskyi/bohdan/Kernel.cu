
#include <hip/hip_runtime.h>
extern "C"
__global__ void matrixAdd(int n, int *a, int *b, int *c) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	int index = i + j * n;
	c[index] = (a[index] + b[index]) * 5 + 8;
}