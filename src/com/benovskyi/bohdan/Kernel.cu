#include "hip/hip_runtime.h"
extern "C"
__global__ void matrixAdd(int *a, int *b, int *c) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	int index = i + j * N;
	c[index] = (a[index] + b[index]) * 5 + 8;
}